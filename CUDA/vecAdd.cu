#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 



typedef float Num;
typedef int cantidad;
// CUDAKernel. Cada Thread tomara un elemento de C

int tamano_bloque, tamano_grid;

__global__ void vecAddKernel(Num *A, Num *B, Num *C, int n)
{
    // Obtener el Id del tread
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < n) C[id] = A[id] + B[id];
}
 
void vecAdd(Num* A, Num* B,Num* C,cantidad n){
	int size=n*sizeof(Num);
	Num *d_A, *d_B, *d_C;
	hipMalloc((void**) &d_A, size);
    hipMemcpy( d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_B, size);
    hipMemcpy( d_B, B, size, hipMemcpyHostToDevice);
    
    hipMalloc((void**) &d_C, size);

    tamano_grid = (cantidad)ceil((float)n/tamano_bloque);
	
	hipEventRecord(start);
	vecAddKernel<<<tamano_grid, tamano_bloque>>>(d_A, d_B, d_C, n);
	hipEventRecord(stop);   
    
 	hipEventSynchronize(stop);
	float milisec = 0;
	hipEventElapsedTime(&milisec, start, stop);

	printf("%f\n", milisec );
    hipMemcpy( C, d_C, size, hipMemcpyDeviceToHost );
    hipFree(d_A);hipFree(d_B);hipFree(d_C);

}

int main( int argc, char** argv )
{
	if(argc != 3){
		printf("Faltan Argumentos <blockSize> <tamVector>\n");
		return 0;
	}    
    int n = atoi(argc[2]);;
 	tamano_bloque=atoi(argc[1]);
    
    Num *A;
    Num *B;
    
    Num *C;
	
	size_t size = n*sizeof(Num);
 
    A = (Num*)malloc(size);
    B = (Num*)malloc(size);
    C = (Num*)malloc(size);
 
    int i;
    
    for( i = 0; i < n; i++ ) {
        A[i] = sin(i)*sin(i);
        B[i] = cos(i)*cos(i);
    }
    vecAdd(A,B,C,n);
 
    Num sum = 0;
    for(i=0; i<n; i++)
        sum += C[i];
    printf("final result: %f\n", sum/n);
 
    free(A);
    free(B);
    free(C);
 
    return 0;
}
